#include "cuda_kernel.cuh"

// ====================================================================
// create context, it contains all data that gpu needs to use
// ====================================================================
context* init_context()
{
    hipSetDevice(1);
    context* ctx;
    
    hipMallocManaged((void**)&ctx, sizeof(context));

    ctx->resolution[0] = DIM_X;
    ctx->resolution[1] = DIM_Y;
    ctx->resolution[2] = DIM_Z;

    ctx->trunc_margin = 5 * VOXEL_SIZE;

    ctx->weight_threshhold = WEIGHT_THRESHOLD;
    ctx->L0_voxel_num = 0;
    ctx->L2_voxel_num = 0;

    int voxel_num = ctx->resolution[0] * ctx->resolution[1] * ctx->resolution[2];

    hipMalloc((void**)&ctx->tsdf_voxel, voxel_num * sizeof(float));
    hipMalloc((void**)&ctx->color_voxel, voxel_num * sizeof(uint8_t) * 3);
    hipMalloc((void**)&ctx->valid_voxel, voxel_num * sizeof(baseVoxel));
    hipMalloc((void**)&ctx->in_buf_depth, CAM_NUM * WIDTH * HEIGHT * sizeof(uint8_t));
    hipMalloc((void**)&ctx->in_buf_color, CAM_NUM * WIDTH * HEIGHT * sizeof(uint8_t) * 3);
    hipMalloc((void**)&ctx->depth, CAM_NUM * WIDTH * HEIGHT * sizeof(float));
    hipMalloc((void**)&ctx->pcd, 3 * WIDTH * HEIGHT * sizeof(float));

    hipMemset(ctx->tsdf_voxel, 1, voxel_num * sizeof(float));
    hipMemset(ctx->color_voxel, 0, voxel_num * sizeof(uint8_t) * 3);
    HANDLE_ERROR();

    return ctx;
}


// ====================================================================
// dequantization is necessary since depth has been quantizated to 0-255
// ====================================================================
void dequantization(context* ctx, uint8_t *input_depth, float *output_depth)
{
    int width = ctx->width;
    int height = ctx->height;

    dim3 blocks(width / 32, height / 24);
    dim3 threads(32, 24);

    hipMemset(output_depth, 0, width * height * sizeof(float));
    dequantization_kernel<<<blocks, threads>>>(ctx, input_depth, output_depth);
}


// ====================================================================
// core function, integrate an depth frame to volume
// ====================================================================
void Integrate(context* ctx, uint8_t *in_buf_depth, uint8_t* in_buf_color)
{
#ifdef TimeEventRecord
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
#endif
    reset_context(ctx);
    hipMemcpy(ctx->in_buf_depth, in_buf_depth, CAM_NUM * WIDTH * HEIGHT * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(ctx->in_buf_color, in_buf_color, CAM_NUM * 3 * WIDTH * HEIGHT * sizeof(uint8_t), hipMemcpyHostToDevice);

    for(int i = 0; i < CAM_NUM; i++) {
        dequantization(ctx, ctx->in_buf_depth + WIDTH * HEIGHT * i, ctx->depth + WIDTH * HEIGHT * i);
        HANDLE_ERROR();
    }

    Lock *lock;
    hipMallocManaged((void**)&lock, sizeof(Lock));

    integrate_L0_kernel<<<dim3(DIM_Z / 32, DIM_Y / 32), dim3(32, 32)>>>(ctx, lock);
    hipDeviceSynchronize(); // force cpu to wait util kernel finish

    int block_config = (ctx->L0_voxel_num + 255) / 256;
    integrate_L1_kernel<<<block_config, 256>>>(ctx, lock);
    
    HANDLE_ERROR();

#ifdef TimeEventRecord
    hipEventRecord(end);
    hipEventSynchronize(end);
    float millisecond = 0;
    hipEventElapsedTime(&millisecond, start, end);
    printf("\t Integrate time = %f ms\n", millisecond);
#endif
}


void get_pcd_in_world(context* ctx, uint8_t *in_buf_depth, float *pcd, int cam_idx)
{
    hipMemcpy(ctx->in_buf_depth, in_buf_depth, WIDTH * HEIGHT * sizeof(uint8_t), hipMemcpyHostToDevice);
    dequantization(ctx, ctx->in_buf_depth, ctx->depth);

    dim3 blocks(WIDTH / 32, HEIGHT / 24);
    dim3 threads(32, 24);

    depth_to_world_pcd<<<blocks, threads>>>(ctx, cam_idx);

    hipMemcpy(pcd, ctx->pcd, 3 * WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
}


void memcpy_volume_to_cpu(context* ctx, baseVoxel* voxel_out, int &voxel_num)
{
    voxel_num = ctx->L0_voxel_num + ctx->L1_voxel_num + ctx->L2_voxel_num;
    voxel_num = ctx->L1_voxel_num;
    hipMemcpy(voxel_out, ctx->valid_voxel + ctx->L0_voxel_num, voxel_num * sizeof(baseVoxel), hipMemcpyDeviceToHost);
}


void reset_context(context* ctx)
{
    ctx->L0_voxel_num = 0;
    ctx->L1_voxel_num = 0;
    ctx->L2_voxel_num = 0;
}


// ====================================================================
// release memory in GPU
// ====================================================================
void release_context(context* ctx)
{
    hipFree(ctx->tsdf_voxel);
    hipFree(ctx->color_voxel);
    hipFree(ctx->valid_voxel);
    hipFree(ctx->in_buf_depth);
    hipFree(ctx->in_buf_color);
    hipFree(ctx->depth);
    hipFree(ctx->pcd);
    hipFree(ctx);
}
